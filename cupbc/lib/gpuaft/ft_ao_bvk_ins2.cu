/*
Copyright (c) 2024 Bytedance Ltd. and/or its affiliates
This file is part of ByteQC.

ByteQC is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

ByteQC is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_complex.h>
#include <assert.h>
#include "ft_ao.h"

#include "ft_ao_template.cu"

template void _PBC_ft_bvk<true, 5, 5>(const int, hipDoubleComplex *, const int,
    const int, const int, const int, const int *, const int8_t *, double *,
    const double *, const hipDoubleComplex *, const int *, const int *,
    const double *, const double *, const int *, const int *, const int, int *,
    const int, int *, const int);
